#include "hip/hip_runtime.h"


#include <stdio.h>

#define SIZE 64

__global__ void vectorAddSub(int* A, int* B, int* C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int warpIdValue = 0;
    warpIdValue = threadIdx.x / 32;

    if (warpIdValue % 2 == 0) {
        C[i] = A[i] + B[i];
    } else {
        C[i] = A[i] - B[i];
    }
}

int main()
{
    int* A, * B, * C;
    int* dA, * dB, * dC;
    int size = SIZE * sizeof(int);

    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);

    hipMalloc((void**)&dA, size);
    hipMalloc((void**)&dB, size);
    hipMalloc((void**)&dC, size);

    for (int i = 0; i < SIZE; i++)
    {
        A[i] = i;
        B[i] = i;
    }
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, size, hipMemcpyHostToDevice);

    vectorAddSub <<<2, 64>>> (dA, dB, dC, SIZE);

    hipMemcpy(C, dC, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE/2+1; i+=2)
    {
        printf("%d + %d = %d\n", A[i], B[i], C[i]);
    }

    for (int i = 1; i < SIZE/2; i+=2)
    {
        printf("%d - %d = %d\n", A[i], B[i], C[i]);
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(A);
    free(B);
    free(C);

    return 0;
}

