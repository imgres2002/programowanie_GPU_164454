#include "hip/hip_runtime.h"
#include <iostream>

#define SIZE 4100
#define CHUNK_SIZE 1024

// kernel sumujacy wektory element po elemencie
__global__ void vectorAdd(int* A, int* B, int* C, int n)
{
    int i = threadIdx.x;
    if (i >= n) {
        return;
    }
    C[i] = A[i] + B[i];
}

int main()
{
    // krok 1. alokacja przestrzeni w pamięci RAM CPU
    int* A, * B, * C;
    int* dA, * dB, * dC;
    int size = SIZE * sizeof(int);
    int chunkSize = CHUNK_SIZE * sizeof(int);

    // krok 2. alokacja wektorów w pamięci RAM CPU
    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);

    // krok 3. alokacja pamięci RAM GPU
    hipMalloc((void**)&dC, chunkSize);
    hipMalloc((void**)&dA, chunkSize);
    hipMalloc((void**)&dB, chunkSize);

    // krok 4. inicjalizacja wartości wektorów
    for (int i = 0; i < SIZE; i++)
    {
        A[i] = i;
        B[i] = SIZE - i;
    }

    for (int i = 0; i < SIZE-CHUNK_SIZE; i += CHUNK_SIZE) {
        // krok 5. transfer wektorów z pamięci RAM CPU do pamięci GPU
        hipMemcpy(dA, &A[i], chunkSize, hipMemcpyHostToDevice);
        hipMemcpy(dB, &B[i], chunkSize, hipMemcpyHostToDevice);

        // krok 6. uruchomienie kernela
        int blocks = (CHUNK_SIZE + 1024 - 1) / 1024;
        vectorAdd <<<blocks, 1024>>> (dA, dB, dC, CHUNK_SIZE);
        hipDeviceSynchronize();

        // krok 7. transfer wektora wynikowego z pamięci GPU do CPU
        hipMemcpy(&C[i], dC, chunkSize, hipMemcpyDeviceToHost);
    }
    int reminder = SIZE % CHUNK_SIZE;
    int last_chunk = SIZE - reminder;
    hipMemcpy(dA, &A[last_chunk], chunkSize, hipMemcpyHostToDevice);
    hipMemcpy(dB, &B[last_chunk], chunkSize, hipMemcpyHostToDevice);
    int blocks = (CHUNK_SIZE + 1024 - 1) / 1024;
    vectorAdd <<<blocks, 1024>>> (dA, dB, dC, reminder);
    hipDeviceSynchronize();
    hipMemcpy(&C[last_chunk], dC, reminder * sizeof(int), hipMemcpyDeviceToHost);

    // krok 8. zwrócenie wyników do wyjścia standardowego
    for (int i = 0; i < SIZE; i++)
    {
        printf("%d + %d = %d\n", A[i], B[i], C[i]);
    }

    // krok 9. wyczyszczenie pamięci GPU oraz CPU
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(A);
    free(B);
    free(C);

    return 0;
}
