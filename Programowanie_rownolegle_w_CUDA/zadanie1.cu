#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void kernel() // kernel, to be executed on GPU
{
	printf("\nThe thread ID is %d", threadIdx.x);
}

int main() // function, execute on CPU
{
	// format: <<<num_of_blocks, num_of_threads_per_block>>>
	kernel << <2, 128 >> > ();

	return 0;
}