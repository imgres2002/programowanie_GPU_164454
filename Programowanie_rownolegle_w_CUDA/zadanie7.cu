#include "hip/hip_runtime.h"


#include <stdio.h>

#define SIZE 2048

__global__ void vectorAddSub(int* A, int* B, int* C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int i_even = 2 * i;
    if (i_even < n) {
        C[i_even] = A[i_even] + B[i_even];
    }

    int i_odd = 2 * i + 1;
    if (i_odd < n) {
        C[i_odd] = A[i_odd] - B[i_odd];
    }
}

int main()
{
    int* A, * B, * C;
    int* dA, * dB, * dC;
    int size = SIZE * sizeof(int);

    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);

    hipMalloc((void**)&dA, size);
    hipMalloc((void**)&dB, size);
    hipMalloc((void**)&dC, size);

    for (int i = 0; i < SIZE; i++)
    {
        A[i] = i;
        B[i] = i;
    }
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, size, hipMemcpyHostToDevice);

    vectorAddSub <<<64, 32>>> (dA, dB, dC, SIZE);

    hipMemcpy(C, dC, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE/2+1; i+=2)
    {
        printf("%d + %d = %d\n", A[i], B[i], C[i]);
    }

    for (int i = 1; i < SIZE/2; i+=2)
    {
        printf("%d - %d = %d\n", A[i], B[i], C[i]);
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(A);
    free(B);
    free(C);

    return 0;
}
