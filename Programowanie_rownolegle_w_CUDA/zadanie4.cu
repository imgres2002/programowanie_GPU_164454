#include "hip/hip_runtime.h"


#include <stdio.h>

#define SIZE 2048

__global__ void vectorAdd(int* A, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    A[i] += threadIdx.x + blockIdx.x;
}

int main()
{
    int* A;
    int* dA;
    int size = SIZE * sizeof(int);

    A = (int*)malloc(size);

    hipMalloc((void**)&dA, size);

    for (int i = 0; i < SIZE; i++) {
        A[i] = 100;
    }
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);

    vectorAdd <<<64, 32>>> (dA, SIZE);

    hipMemcpy(A, dA, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE; i++) {
        printf("%d\n", A[i]);
    }

    hipFree(dA);
    free(A);

    return 0;
}
