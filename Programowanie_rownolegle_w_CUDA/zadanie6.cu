#include "hip/hip_runtime.h"


#include <stdio.h>

#define SIZE 2048

__global__ void vectorSubtract(int* A, int* B, int* C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    C[i] = A[i] - B[i];
}

int main()
{
    int* A, * B, * C;
    int* dA, * dB, * dC;
    int size = SIZE * sizeof(int);

    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);

    hipMalloc((void**)&dA, size);
    hipMalloc((void**)&dB, size);
    hipMalloc((void**)&dC, size);

    for (int i = 0; i < SIZE; i++)
    {
        A[i] = i;
        B[i] = i;
    }
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, size, hipMemcpyHostToDevice);

    vectorSubtract <<<64, 32>>> (dA, dB, dC, SIZE);

    hipMemcpy(C, dC, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE; i++) {
        printf("%d - %d = %d\n", A[i], B[i], C[i]);
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(A);
    free(B);
    free(C);

    return 0;
}
